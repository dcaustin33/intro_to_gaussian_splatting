#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <cmath>  // Include this header for expf function
#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdexcept>

#include <pybind11/pybind11.h>

// #define PRINT_DEBUG
#define TILE_SIZE 16
#define CUDA_CHECK(call)                                                                   \
    {                                                                                      \
        hipError_t err = call;                                                            \
        if (err != hipSuccess)                                                            \
        {                                                                                  \
            printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            throw std::runtime_error(hipGetErrorString(err));                             \
        }                                                                                  \
    }
#define CHECK_CUDA_INPUT(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS_INPUT(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x)   \
    CHECK_CUDA_INPUT(x); \
    CHECK_CONTIGUOUS_INPUT(x)

namespace py = pybind11;

__global__ void get_start_idx_kernel(
    float* array,
    int* starting_idx,
    int total_x_tiles,
    int total_y_tiles,
    int array_length
)
{
    // we are only going to use 1d blocks and 1d thread blocks
    int array_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (array_idx >= array_length)
    {
        return;
    }

    int tile_x = static_cast<int>(array[4 * array_idx]);
    int tile_y = static_cast<int>(array[4 * array_idx + 1]);

    // Compute linear index into array_map
    int map_idx = tile_y * total_x_tiles + tile_x;
    int *ptr = &starting_idx[map_idx];

    // Try to set this position if it's -1
    int old_val = atomicCAS(ptr, -1, array_idx);

    if (old_val != -1) {
        // If the slot wasn't -1, we only update if idx < old_val
        // old_val here is what was previously at *ptr before CAS
        // We must re-check the current value in *ptr, since atomicCAS 
        // could have changed it if another thread updated in between.
        int cur_val = atomicAdd(ptr, 0); // atomicAdd with 0 to read atomically
        if (array_idx < cur_val) {
            // Use atomicMin to attempt to reduce the value
            atomicMin(ptr, array_idx);
        }
    }
}


torch::Tensor get_start_idx_cuda(
    torch::Tensor array,
    int total_x_tiles,
    int total_y_tiles
)
{
    CHECK_INPUT(array);
    torch::Tensor starting_idx = torch::zeros({total_y_tiles, total_x_tiles}, torch::TensorOptions().dtype(torch::kInt32).device(array.device()));
    int array_length = array.size(0);
    dim3 grid_size((array_length + TILE_SIZE*TILE_SIZE - 1) / (TILE_SIZE*TILE_SIZE), 1);
    dim3 block_size(TILE_SIZE * TILE_SIZE, 1);
    get_start_idx_kernel<<<grid_size, block_size>>>(
        array.data_ptr<float>(),
        starting_idx.data_ptr<int>(),
        total_x_tiles,
        total_y_tiles,
        array_length
    );
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    return starting_idx;
}

PYBIND11_MODULE(preprocessing, m)
{
    m.def("get_start_idx_cuda", &get_start_idx_cuda, "Get the start idx of the tile");
}