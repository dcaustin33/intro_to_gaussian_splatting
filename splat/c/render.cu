#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath> // Include this header for expf function
#include <torch/extension.h>

__device__ float compute_pixel_strength(
    int pixel_x,
    int pixel_y,
    int point_x,
    int point_y,
    float inverse_covariance_a,
    float inverse_covariance_b,
    float inverse_covariance_c)
{
    // Compute the distance between the pixel and the point
    float dx = pixel_x - point_x;
    float dy = pixel_y - point_y;
    float power = dx * inverse_covariance_a * dx + 2 * dx * dy * inverse_covariance_b + dy * dy * inverse_covariance_c;
    return expf(-0.5f * power);
}

__global__ void render_tile(
    int image_height,
    int image_width,
    int tile_size,
    int num_points,
    float *point_means,
    float *point_colors,
    float *image,
    float *inverse_covariance_2d,
    float *min_x,
    float *max_x,
    float *min_y,
    float *max_y,
    float *opacity)
{
    // Calculate the pixel's position in the image
    int pixel_x = blockIdx.x * tile_size + threadIdx.x;
    int pixel_y = blockIdx.y * tile_size + threadIdx.y;

    // Ensure the pixel is within the image bounds
    if (pixel_x >= image_width || pixel_y >= image_height)
    {
        return;
    }

    float total_weight = 1.0f;
    float3 color = {0.0f, 0.0f, 0.0f};

    for (int i = 0; i < num_points; i++)
    {
        float point_x = point_means[i * 2];
        float point_y = point_means[i * 2 + 1];

        // checks to make sure we are within the bounding box
        bool x_check = pixel_x >= min_x[i] && pixel_x <= max_x[i];
        bool y_check = pixel_y >= min_y[i] && pixel_y <= max_y[i];
        if (!x_check || !y_check)
        {
            continue;
        }
        float strength = compute_pixel_strength(
            pixel_x,
            pixel_y,
            point_x,
            point_y,
            inverse_covariance_2d[i * 4],
            inverse_covariance_2d[i * 4 + 1],
            inverse_covariance_2d[i * 4 + 3]);
        
        float initial_alpha = opacity[i] * strength;
        float alpha = min(.99f, initial_alpha);
        float test_weight = total_weight * (1 - alpha);
        if (test_weight < 0.001f)
        {
            break;
        }
        color.x += total_weight * alpha * point_colors[i * 3];
        color.y += total_weight * alpha * point_colors[i * 3 + 1];
        color.z += total_weight * alpha * point_colors[i * 3 + 2];
        total_weight = test_weight;
    }

    image[(pixel_y * image_width + pixel_x) * 3] = color.x;
    image[(pixel_y * image_width + pixel_x) * 3 + 1] = color.y;
    image[(pixel_y * image_width + pixel_x) * 3 + 2] = color.z;

}


torch::Tensor render_image(
    int image_height,
    int image_width,
    int tile_size,
    torch::Tensor point_means,
    torch::Tensor point_colors,
    torch::Tensor inverse_covariance_2d,
    torch::Tensor min_x,
    torch::Tensor max_x,
    torch::Tensor min_y,
    torch::Tensor max_y,
    torch::Tensor opacity)
{
    // Ensure the input tensors are on the same device
    torch::TensorArg point_means_t{point_means, "point_means", 1},
        point_colors_t{point_colors, "point_colors", 2},
        inverse_covariance_2d_t{inverse_covariance_2d, "inverse_covariance_2d", 3},
        min_x_t{min_x, "min_x", 4},
        max_x_t{max_x, "max_x", 5},
        min_y_t{min_y, "min_y", 6},
        max_y_t{max_y, "max_y", 7},
        opacity_t{opacity, "opacity", 8};
    torch::checkAllSameGPU("render_image", {point_means_t, point_colors_t, inverse_covariance_2d_t, min_x_t, max_x_t, min_y_t, max_y_t, opacity_t});

    
    // Create an output tensor for the image
    torch::Tensor image = torch::zeros({image_height, image_width, 3}, point_means.options());

    // Calculate the number of tiles in the image
    int num_tiles_x = (image_width + tile_size - 1) / tile_size;
    int num_tiles_y = (image_height + tile_size - 1) / tile_size;

    // Launch a CUDA kernel to render the image
    dim3 block(tile_size, tile_size);
    dim3 grid(num_tiles_x, num_tiles_y);
    render_tile<<<grid, block>>>(
        image_height,
        image_width,
        tile_size,
        point_means.size(0),
        point_means.data_ptr<float>(),
        point_colors.data_ptr<float>(),
        image.data_ptr<float>(),
        inverse_covariance_2d.data_ptr<float>(),
        min_x.data_ptr<float>(),
        max_x.data_ptr<float>(),
        min_y.data_ptr<float>(),
        max_y.data_ptr<float>(),
        opacity.data_ptr<float>());

    return image;
}