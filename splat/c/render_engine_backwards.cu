#include "hip/hip_runtime.h"
/*
This is a copy of the forwards pass where I will add backwards components step by step
*/

#include <torch/extension.h>

#include <cmath>  // Include this header for expf function
#include <cstdio>
#include <hip/hip_runtime.h>

#include <pybind11/pybind11.h>

// #define PRINT_DEBUG 1
#define TILE_SIZE 16
#define CUDA_CHECK(call)                                                                   \
    {                                                                                      \
        hipError_t err = call;                                                            \
        if (err != hipSuccess)                                                            \
        {                                                                                  \
            printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            throw std::runtime_error(hipGetErrorString(err));                             \
        }                                                                                  \
    }
#define CHECK_CUDA_INPUT(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS_INPUT(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x)   \
    CHECK_CUDA_INPUT(x); \
    CHECK_CONTIGUOUS_INPUT(x)

void check_inputs(
    torch::Tensor point_means,
    torch::Tensor point_colors,
    torch::Tensor point_opacities,
    torch::Tensor inverse_covariance_2d,
    torch::Tensor image,
    torch::Tensor starting_tile_indices,
    torch::Tensor tile_idx,
    torch::Tensor array_indices,
    torch::Tensor dl_dpixel,
    torch::Tensor dl_dpoint_means,
    torch::Tensor dl_dpoint_colors,
    torch::Tensor dl_dpoint_opacities,
    torch::Tensor dl_dinverted_covariance_2d
)
{
    CHECK_INPUT(point_means);
    CHECK_INPUT(point_colors);
    CHECK_INPUT(point_opacities);
    CHECK_INPUT(inverse_covariance_2d);
    CHECK_INPUT(image);
    CHECK_INPUT(starting_tile_indices);
    CHECK_INPUT(tile_idx);
    CHECK_INPUT(array_indices);
}

namespace py = pybind11;

__device__ float sigmoid(float x)
{
    return 1.0f / (1.0f + exp(-x));
}

__device__ float compute_pixel_strength(
    int pixel_x,
    int pixel_y,
    float point_x,
    float point_y,
    float inverse_covariance_a,
    float inverse_covariance_b,
    float inverse_covariance_c)
{
    // Compute the distance between the pixel and the point
    float dx = pixel_x - point_x;
    float dy = pixel_y - point_y;

    float power = dx * inverse_covariance_a * dx +
                  2 * dx * dy * inverse_covariance_b +
                  dy * dy * inverse_covariance_c;
    if (power < 0)
    {
        // according to chatgpt indicates numerical
        // instability as this should never occur
        return 0.0f;
    }
    return exp(-0.5f * power);
}

__global__ void render_tile_kernel_backwards(
    int tile_size,
    float* point_means,
    float* point_colors,
    float* point_opacities,
    float* inverse_covariance_2d,
    float* image,
    int* starting_tile_indices,
    int* tile_idx,
    int* array_indices,
    int image_height,
    int image_width,
    int num_points,
    int num_array_points,
    float* dl_dpixel,
    float* dl_dmeans,
    float* dl_dcolors,
    float* dl_dopacities,
    float* dl_dinverted_covariance_2d
    )
{
    if (tile_size != TILE_SIZE)
    {
        printf("Error: Tile size must be %d but got %d\n", TILE_SIZE, tile_size);
        return;
    }
    // so we need to load all the points
    // then each will have shared memory corresponding to
    // means, color, opacity, covariance, and then the tile id
    int tile_x = blockIdx.x;
    int tile_y = blockIdx.y;
    int pixel_x = threadIdx.x + blockIdx.x * tile_size;
    int pixel_y = threadIdx.y + blockIdx.y * tile_size;
    bool done = false;

    __shared__ float shared_point_means[TILE_SIZE * TILE_SIZE * 2];
    __shared__ bool shared_done_indicator[TILE_SIZE * TILE_SIZE];
    __shared__ float shared_point_colors[TILE_SIZE * TILE_SIZE * 3];
    __shared__ float shared_point_opacities[TILE_SIZE * TILE_SIZE];
    __shared__ float shared_inverse_covariance_2d[TILE_SIZE * TILE_SIZE * 3];
    __shared__ int shared_gaussian_idx[TILE_SIZE * TILE_SIZE];

    float dl_dalpha[TILE_SIZE * TILE_SIZE];

    if (pixel_x >= image_width || pixel_y >= image_height)
    {
        // still helps with the shared memory so no return
        done = true;
    }

    int target_pixel_x = 17;
    int target_pixel_y = 16;
    int target_tile_x = target_pixel_x / TILE_SIZE;
    int target_tile_y = target_pixel_y / TILE_SIZE;

#ifdef PRINT_DEBUG
    if (tile_x != target_tile_x && tile_y != target_tile_y)
    {
        return;
    }
#endif

    // then we have to load and if their tile does not match we indicate done in
    // the array
    int thread_dim = blockDim.x * blockDim.y;
    int round_counter = 0;
    int point_idx;
    float total_weight = 1.0f;
    float3 color = {0.0f, 0.0f, 0.0f};
    int num_done = 0;

    int correct_tile_idx = tile_y * gridDim.x + tile_x;
    int thread_idx = threadIdx.x + threadIdx.y * blockDim.x;
    shared_done_indicator[thread_idx] = false;

    if (starting_tile_indices[correct_tile_idx] == -1)
    {
        return;
    }
    int pixel_idx = (pixel_y * image_width + pixel_x) * 3;

    while (true)
    {
        __syncthreads();
        shared_done_indicator[thread_idx] = false;
        num_done = __syncthreads_count(done);
        if (num_done == thread_dim)
            break;

        // Calculate global point index for this round
        point_idx = starting_tile_indices[correct_tile_idx] + round_counter * thread_dim;
        if (point_idx < 0)
        {
            shared_done_indicator[thread_idx] = true;
        }

        // Calculate global memory offset for this point
        int point_offset = point_idx + thread_idx;
        if (point_offset >= num_array_points)
        {
            shared_done_indicator[thread_idx] = true;
        } else
        {
            int processed_gaussians_idx = array_indices[point_offset];
            if (processed_gaussians_idx >= num_points || processed_gaussians_idx < 0)
            {
                shared_done_indicator[thread_idx] = true;
            } else
            {
                // Load point data into shared memory
                shared_point_means[thread_idx * 2] = point_means[processed_gaussians_idx * 3];
                shared_point_means[thread_idx * 2 + 1] = point_means[processed_gaussians_idx * 3 + 1];
                shared_point_colors[thread_idx * 3] = point_colors[processed_gaussians_idx * 3];
                shared_point_colors[thread_idx * 3 + 1] = point_colors[processed_gaussians_idx * 3 + 1];
                shared_point_colors[thread_idx * 3 + 2] = point_colors[processed_gaussians_idx * 3 + 2];
                shared_point_opacities[thread_idx] = point_opacities[processed_gaussians_idx];

                // no need to store the other covariance as its symmetric
                shared_inverse_covariance_2d[thread_idx * 3] = inverse_covariance_2d[processed_gaussians_idx * 4];
                shared_inverse_covariance_2d[thread_idx * 3 + 1] = inverse_covariance_2d[processed_gaussians_idx * 4 + 1];
                shared_inverse_covariance_2d[thread_idx * 3 + 2] = inverse_covariance_2d[processed_gaussians_idx * 4 + 3];
                shared_gaussian_idx[thread_idx] = processed_gaussians_idx;
            }
            if (tile_idx[point_offset] != correct_tile_idx)
            {
                shared_done_indicator[thread_idx] = true;
            }
        }

        // wait for all the memory loads to finish
        __syncthreads();
        round_counter++;
        int shared_done_count = 0;
        if (!done)
        {
            // render the pixel by iterating through all points until weight or
            // a done indicator is reached
            for (int i = 0; i < thread_dim; i++)
            {
                if (shared_done_indicator[i])
                {
                    shared_done_count++;
                    continue;
                } else
                {
                    float gaussian_strength = compute_pixel_strength(
                        pixel_x,
                        pixel_y,
                        shared_point_means[i * 2],
                        shared_point_means[i * 2 + 1],
                        shared_inverse_covariance_2d[i * 3],
                        shared_inverse_covariance_2d[i * 3 + 1],
                        shared_inverse_covariance_2d[i * 3 + 2]);

                    float opacity_output = sigmoid(shared_point_opacities[i]);
                    float alpha_value = min(0.99f, gaussian_strength * opacity_output);
#ifdef PRINT_DEBUG
                    if (pixel_x == target_pixel_x && pixel_y == target_pixel_y)
                    {
                        printf("alpha_value: %f\n", alpha_value);
                    }
#endif
                    if (alpha_value < 1.0f / 255.0f)
                    {
                        continue;
                    }
                    float test_T = total_weight * (1.0f - alpha_value);
#ifdef PRINT_DEBUG
                    if (target_pixel_x == pixel_x && target_pixel_y == pixel_y)
                    {
                        printf("test_T: %f, gaussian_strength: %f, alpha: %f, mean1: %f, mean2: %f, opacity: %f, color: %f, %f, %f\n",
                               test_T, gaussian_strength, alpha_value,
                               shared_point_means[i * 2], shared_point_means[i * 2 + 1],
                               opacity_output,
                               shared_point_colors[i * 3], shared_point_colors[i * 3 + 1], 
                               shared_point_colors[i * 3 + 2]);
                    }
#endif
                    if (test_T < 0.0001f)
                    {
                        done = true;
                        continue;
                    }
                    color.x += total_weight * alpha_value * shared_point_colors[i * 3];
                    color.y += total_weight * alpha_value * shared_point_colors[i * 3 + 1];
                    color.z += total_weight * alpha_value * shared_point_colors[i * 3 + 2];
                    float current_T = total_weight;
                    total_weight = test_T;
                    // TODO: this is where the backwards pass will happen as we know it has contributed
                    int gaussian_idx = shared_gaussian_idx[i];

#ifdef PRINT_DEBUG
                    if (target_pixel_x == pixel_x && target_pixel_y == pixel_y)
                    {
                        printf("gaussian_idx: %d\n", gaussian_idx);
                        printf("dl_dcolors: %f, %f, %f\n", dl_dcolors[gaussian_idx * 3], dl_dcolors[gaussian_idx * 3 + 1], dl_dcolors[gaussian_idx * 3 + 2]);
                        printf("dl_dpixel: %f, %f, %f\n", dl_dpixel[pixel_idx + 0], dl_dpixel[pixel_idx + 1], dl_dpixel[pixel_idx + 2]);
                        printf("alpha_value: %f, current_T: %f\n", alpha_value, current_T);
                    }
#endif

                    if (pixel_idx + 2 < image_width * image_height * 3){
                        atomicAdd(&dl_dcolors[gaussian_idx * 3], dl_dpixel[pixel_idx + 0] * alpha_value * current_T);
                        atomicAdd(&dl_dcolors[gaussian_idx * 3 + 1], dl_dpixel[pixel_idx + 1] * alpha_value * current_T);
                        atomicAdd(&dl_dcolors[gaussian_idx * 3 + 2], dl_dpixel[pixel_idx + 2] * alpha_value * current_T);
                    }

                    // // derivate to be used for alpha - this likely does not need to be shared memory
                    // atomicAdd(&dl_dalpha[gaussian_idx], dl_dpixel[pixel_x * 3 + 0] * shared_point_colors[i * 3] * total_weight);
                    // atomicAdd(&dl_dalpha[gaussian_idx], dl_dpixel[pixel_x * 3 + 1] * shared_point_colors[i * 3 + 1] * total_weight);
                    // atomicAdd(&dl_dalpha[gaussian_idx], dl_dpixel[pixel_x * 3 + 2] * shared_point_colors[i * 3 + 2] * total_weight);
                
                    // // derivative wrt unactivated opacity
                    // float derivative_sigmoid = sigmoid(shared_point_opacities[i]) * (1 - sigmoid(shared_point_opacities[i]));
                    // atomicAdd(&dl_dopacities[gaussian_idx], dl_dalpha[gaussian_idx] * derivative_sigmoid * gaussian_strength);
                
                }
            }
            if (shared_done_count == thread_dim)
            {
                // this will eventually cause breaking
                done = true;
            }
        }
    }

#ifdef PRINT_DEBUG
    if (pixel_x == target_pixel_x && pixel_y == target_pixel_y)
    {
        printf("color: %f, %f, %f\n", color.x, color.y, color.z);
    }
#endif
    if (pixel_x < image_width && pixel_y < image_height)
    {
        if (pixel_idx + 2 < image_width * image_height * 3)
        {
            image[pixel_idx] = color.x;
            image[pixel_idx + 1] = color.y;
            image[pixel_idx + 2] = color.z;
        }
    }
}

torch::Tensor render_tile_cuda_backwards(
    int tile_size,
    torch::Tensor point_means,
    torch::Tensor point_colors,
    torch::Tensor point_opacities,
    torch::Tensor inverse_covariance_2d,
    torch::Tensor image,
    torch::Tensor starting_tile_indices,
    torch::Tensor tile_idx,
    torch::Tensor array_indices,
    int image_height,
    int image_width,
    int num_points,
    int num_array_points,
    torch::Tensor dl_dpixel,
    torch::Tensor dl_dpoint_means,
    torch::Tensor dl_dpoint_colors,
    torch::Tensor dl_dpoint_opacities,
    torch::Tensor dl_dinverted_covariance_2d
)
{
    // have to put the tile_size and other ints into tensors to save
    // in the torch autograd
    check_inputs(
        point_means,
        point_colors,
        point_opacities,
        inverse_covariance_2d,
        image,
        starting_tile_indices,
        tile_idx,
        array_indices,
        dl_dpixel,
        dl_dpoint_means,
        dl_dpoint_colors,
        dl_dpoint_opacities,
        dl_dinverted_covariance_2d
    );
    if (tile_size != TILE_SIZE)
    {
        throw std::runtime_error("Tile size must be 16 or TILE_SIZE in c code must change");
    }
    dim3 block_size(tile_size, tile_size);
    int grid_size_x = (image_width + tile_size - 1) / tile_size;
    int grid_size_y = (image_height + tile_size - 1) / tile_size;
    dim3 grid_size(grid_size_x, grid_size_y);

    torch::Tensor image_new = torch::ones({image_height, image_width, 3}, image.options());

    // print the amount of elements in image tensor
    render_tile_kernel_backwards<<<grid_size, block_size>>>(
        tile_size,
        point_means.data_ptr<float>(),
        point_colors.data_ptr<float>(),
        point_opacities.data_ptr<float>(),
        inverse_covariance_2d.data_ptr<float>(),
        image.data_ptr<float>(),
        starting_tile_indices.data_ptr<int>(),
        tile_idx.data_ptr<int>(),
        array_indices.data_ptr<int>(),
        image_height,
        image_width,
        num_points,
        num_array_points,
        dl_dpixel.data_ptr<float>(),
        dl_dpoint_means.data_ptr<float>(),
        dl_dpoint_colors.data_ptr<float>(),
        dl_dpoint_opacities.data_ptr<float>(),
        dl_dinverted_covariance_2d.data_ptr<float>()
    );
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    return image;
}

PYBIND11_MODULE(render_engine_backwards, m)
{
    m.def("render_tile_cuda_backwards", &render_tile_cuda_backwards, "Perform the backwards pass for the tile");
}